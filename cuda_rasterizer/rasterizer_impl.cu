#include "hip/hip_runtime.h"
/*
 * Copyright (C) 2023, Inria
 * GRAPHDECO research group, https://team.inria.fr/graphdeco
 * All rights reserved.
 *
 * This software is free for non-commercial, research and evaluation use 
 * under the terms of the LICENSE.md file.
 *
 * For inquiries contact  george.drettakis@inria.fr
 */

#include "rasterizer_impl.h"
#include <iostream>
#include <fstream>
#include <algorithm>
#include <numeric>
#include <hip/hip_runtime.h>
#include "hip/hip_runtime.h"
#include ""
#include <hipcub/hipcub.hpp>
#include <cub/device/device_radix_sort.cuh>
#define GLM_FORCE_CUDA
#include <glm/glm.hpp>

#include <hip/hip_cooperative_groups.h>
#include <cooperative_groups/reduce.h>
namespace cg = cooperative_groups;

#include "auxiliary.h"
#include "forward.h"
#include "backward.h"

// Helper function to find the next-highest bit of the MSB
// on the CPU.
//找到输入的32位无符号整数 n 中最高的位（most significant bit，MSB）的位置
uint32_t getHigherMsb(uint32_t n)
{
	uint32_t msb = sizeof(n) * 4;
	uint32_t step = msb;
	while (step > 1)
	{
		step /= 2;
		if (n >> msb)
			msb += step;
		else
			msb -= step;
	}
	if (n >> msb)
		msb++;
	return msb;
}

// Wrapper method to call auxiliary coarse frustum containment test.
// Mark all Gaussians that pass it.
__global__ void checkFrustum(int P,
	const float* orig_points,
	const float* viewmatrix,
	const float* projmatrix,
	bool* present)
{
	auto idx = cg::this_grid().thread_rank();
	if (idx >= P)
		return;

	float3 p_view;
	present[idx] = in_frustum(idx, orig_points, viewmatrix, projmatrix, false, p_view);
}

// Generates one key/value pair for all Gaussian / tile overlaps. 
// Run once per Gaussian (1:N mapping).
__global__ void duplicateWithKeys(
	int P,//点的个数
	const float2* points_xy,//像素坐标
	const float* depths,//深度
	const uint32_t* offsets,//覆盖瓦片的前缀和
	uint64_t* gaussian_keys_unsorted,
	uint32_t* gaussian_values_unsorted,
	int* radii,////三维高斯投影到二维后长轴的值
	dim3 grid)//网格尺寸
{
	auto idx = cg::this_grid().thread_rank();//索引
	if (idx >= P)
		return;

	// Generate no key/value pair for invisible Gaussians
	if (radii[idx] > 0)
	{
		// Find this Gaussian's offset in buffer for writing keys/values.
		uint32_t off = (idx == 0) ? 0 : offsets[idx - 1];//在此之前覆盖的瓦片的数量之和
		uint2 rect_min, rect_max;

		getRect(points_xy[idx], radii[idx], rect_min, rect_max, grid);//相当于知道了该高斯覆盖了哪些瓦片

		// For each tile that the bounding rect overlaps, emit a 
		// key/value pair. The key is |  tile ID  |      depth      |,
		// and the value is the ID of the Gaussian. Sorting the values 
		// with this key yields Gaussian IDs in a list, such that they
		// are first sorted by tile and then by depth. 
		//实现先由瓦片索引，再由深度索引
		for (int y = rect_min.y; y < rect_max.y; y++)
		{
			for (int x = rect_min.x; x < rect_max.x; x++)//遍历覆盖的瓦片
			{
				uint64_t key = y * grid.x + x;//瓦片索引
				key <<= 32;
				key |= *((uint32_t*)&depths[idx]);
				gaussian_keys_unsorted[off] = key;
				gaussian_values_unsorted[off] = idx;
				//这样就相当于gaussian_keys_unsorted和gaussian_values_unsorted分别存储了（瓦片索引和深度值的组合）和（高斯的索引）
				off++;
			}
		}
	}
}

// Check keys to see if it is at the start/end of one tile's range in 
// the full sorted list. If yes, write start/end of this tile. 
// Run once per instanced (duplicated) Gaussian ID.
//把每个瓦片上对应的高斯idx开头结尾存储在range里面
__global__ void identifyTileRanges(int L, uint64_t* point_list_keys, uint2* ranges)
{
	auto idx = cg::this_grid().thread_rank();
	if (idx >= L)
		return;

	// Read tile ID from key. Update start/end of tile range if at limit.
	uint64_t key = point_list_keys[idx];//这是按照深度排序好的
	uint32_t currtile = key >> 32;//当前高斯的瓦片索引
	if (idx == 0)
		ranges[currtile].x = 0;
	else
	{
		uint32_t prevtile = point_list_keys[idx - 1] >> 32;//上一层高斯的瓦片的索引
		if (currtile != prevtile)
		{
			ranges[prevtile].y = idx;
			ranges[currtile].x = idx;
		}
	}
	if (idx == L - 1)
		ranges[currtile].y = L;
		//ranges相当于记录了瓦片上对应高斯idx的开头和结尾
}

// Mark Gaussians as visible/invisible, based on view frustum testing
void CudaRasterizer::Rasterizer::markVisible(
	int P,
	float* means3D,
	float* viewmatrix,
	float* projmatrix,
	bool* present)
{
	checkFrustum << <(P + 255) / 256, 256 >> > (
		P,
		means3D,
		viewmatrix, projmatrix,
		present);
}

CudaRasterizer::GeometryState CudaRasterizer::GeometryState::fromChunk(char*& chunk, size_t P)
{
	GeometryState geom;
	obtain(chunk, geom.depths, P, 128);//这里就是在geom中开辟了好多好多对齐好的空间
	obtain(chunk, geom.clamped, P * 3, 128);
	obtain(chunk, geom.internal_radii, P, 128);
	obtain(chunk, geom.means2D, P, 128);
	obtain(chunk, geom.cov3D, P * 6, 128);
	obtain(chunk, geom.conic_opacity, P, 128);
	obtain(chunk, geom.rgb, P * 3, 128);
	obtain(chunk, geom.tiles_touched, P, 128);
	hipcub::DeviceScan::InclusiveSum(nullptr, geom.scan_size, geom.tiles_touched, geom.tiles_touched, P);
	//由于输入数据指针是nullptr，这个函数调用实际上不会对任何实际数据进行操作。
	//它可能用于初始化或预热设备内存，或者是为了测试和调试目的，确保后续操作的正确性。
	obtain(chunk, geom.scanning_space, geom.scan_size, 128);
	obtain(chunk, geom.point_offsets, P, 128);
	return geom;
}

CudaRasterizer::ImageState CudaRasterizer::ImageState::fromChunk(char*& chunk, size_t N)
{
	ImageState img;
	obtain(chunk, img.accum_alpha, N, 128);
	obtain(chunk, img.n_contrib, N, 128);
	obtain(chunk, img.ranges, N, 128);
	return img;
}

CudaRasterizer::BinningState CudaRasterizer::BinningState::fromChunk(char*& chunk, size_t P)
{
	BinningState binning;
	obtain(chunk, binning.point_list, P, 128);
	obtain(chunk, binning.point_list_unsorted, P, 128);
	obtain(chunk, binning.point_list_keys, P, 128);
	obtain(chunk, binning.point_list_keys_unsorted, P, 128);
	hipcub::DeviceRadixSort::SortPairs(
		nullptr, binning.sorting_size,
		binning.point_list_keys_unsorted, binning.point_list_keys,
		binning.point_list_unsorted, binning.point_list, P);
	obtain(chunk, binning.list_sorting_space, binning.sorting_size, 128);
	return binning;
}

// Forward rendering procedure for differentiable rasterization
// of Gaussians.
// 光栅化前向通路
int CudaRasterizer::Rasterizer::forward(
	std::function<char* (size_t)> geometryBuffer,
	std::function<char* (size_t)> binningBuffer,
	std::function<char* (size_t)> imageBuffer,//三个resizeFunctional
	const int P, int D, int M,//多少个点，球谐系数的层数，球谐系数size(0)个数
	const float* background,//背景颜色
	const int width, int height,
	const float* means3D,//三维点的三维坐标
	const float* shs,//全部的球谐系数
	const float* colors_precomp,//在python中为colors_precomp，输入不是RGB则必须，单纯渲染的时候是空
	const float* opacities,//三维点的不透明度
	const float* scales,//scale参数，暂时不知道是干嘛的，应该和三维高斯有关，维度为N*3（每个点三个数字）
	const float scale_modifier,//应该是尺度参数，默认为1
	const float* rotations,//rotation参数，暂时不知道是干嘛的，应该和三维高斯有关，维度为N*4（每个点四个数字）
	const float* cov3D_precomp,//预先计算好的协方差矩阵，单纯渲染的时候为空，后续计算的
	const float* viewmatrix,//相机外参矩阵
	const float* projmatrix,//投影矩阵，内参矩阵和外参矩阵一通计算得到
	const float* cam_pos,//相机的光心，用位姿反算出来的
	const float tan_fovx, float tan_fovy,//视野角（单侧）
	const bool prefiltered,//默认false
	float* out_color,//输入时还都是0，{3, H, W}
	int* radii,//和三维点size相同，都是0
	bool debug)
{
	const float focal_y = height / (2.0f * tan_fovy);
	const float focal_x = width / (2.0f * tan_fovx);//计算xy的焦距

	size_t chunk_size = required<GeometryState>(P);//调用函数模版，传入模板参数 GeometryState ，返回所需空间+128（？）
	char* chunkptr = geometryBuffer(chunk_size);//根据上面的size，buffer resize之后返回指针
	GeometryState geomState = GeometryState::fromChunk(chunkptr, P);//构造好了geom（geomState）所需的空间
	//存储一堆关于高斯的内容

	if (radii == nullptr)
	{
		radii = geomState.internal_radii;
	}

	dim3 tile_grid((width + BLOCK_X - 1) / BLOCK_X, (height + BLOCK_Y - 1) / BLOCK_Y, 1);
	//网格尺寸，需要多少个线程块
	dim3 block(BLOCK_X, BLOCK_Y, 1);
	//线程块尺寸

	// Dynamically resize image-based auxiliary buffers during training
	size_t img_chunk_size = required<ImageState>(width * height);
	char* img_chunkptr = imageBuffer(img_chunk_size);//和上面一样
	ImageState imgState = ImageState::fromChunk(img_chunkptr, width * height);//构造了imgstate所需的空间
	//存储图片内容

	if (NUM_CHANNELS != 3 && colors_precomp == nullptr)
	{
		throw std::runtime_error("For non-RGB, provide precomputed Gaussian colors!");
		//非rgb必须提供colors_precomp
	}

	// Run preprocessing per-Gaussian (transformation, bounding, conversion of SHs to RGB)
	// 预处理
	// CHECK_CUDA检查cuda操作是否成功
	CHECK_CUDA(FORWARD::preprocess(
		P, D, M,//多少个点，球谐系数的层数，球谐系数size(0)个数
		means3D,//三维点的三维坐标
		(glm::vec3*)scales,//scale参数，暂时不知道是干嘛的，应该和三维高斯有关，维度为N*3（每个点三个数字）
		scale_modifier,//应该是尺度参数，默认为1
		(glm::vec4*)rotations,//rotation参数，暂时不知道是干嘛的，应该和三维高斯有关，维度为N*4（每个点四个数字）
		opacities,//三维点的不透明度
		shs,//全部的球谐系数
		geomState.clamped,
		cov3D_precomp,//预先计算好的协方差矩阵，单纯渲染的时候为空，后续计算的
		colors_precomp,//在python中为colors_precomp，输入不是RGB则必须，单纯渲染的时候是空
		viewmatrix, projmatrix,//相机外参矩阵；投影矩阵，内参矩阵和外参矩阵一通计算得到
		(glm::vec3*)cam_pos,//相机的光心，用位姿反算出来的
		width, height,
		focal_x, focal_y,//xy的焦距
		tan_fovx, tan_fovy,
		radii,//和三维点size相同，都是0
		geomState.means2D,//像素坐标
		geomState.depths,
		geomState.cov3D,
		geomState.rgb,
		geomState.conic_opacity,
		tile_grid,//网格尺寸
		geomState.tiles_touched,//覆盖网格的面积
		prefiltered//默认false
	), debug)

	// Compute prefix sum over full list of touched tile counts by Gaussians
	// E.g., [2, 3, 0, 2, 1] -> [2, 5, 5, 7, 8]、
	//用于在 CUDA 设备上执行包含所有输入元素的前缀和（inclusive prefix sum）计算
	//将覆盖瓦片数量的和存储在point_offsets中
	CHECK_CUDA(hipcub::DeviceScan::InclusiveSum(geomState.scanning_space, geomState.scan_size, geomState.tiles_touched, geomState.point_offsets, P), debug)

	// Retrieve total number of Gaussian instances to launch and resize aux buffers
	int num_rendered;
	CHECK_CUDA(hipMemcpy(&num_rendered, geomState.point_offsets + P - 1, sizeof(int), hipMemcpyDeviceToHost), debug);
	// 把指针（point_offsets + P - 1），也就是point_offsets数组的最后一个元素的值，赋给num_rendered，也就是总共覆盖的tiles数量
	size_t binning_chunk_size = required<BinningState>(num_rendered);
	char* binning_chunkptr = binningBuffer(binning_chunk_size);
	BinningState binningState = BinningState::fromChunk(binning_chunkptr, num_rendered);//与上面一样，resize后返回指针

	// For each instance to be rendered, produce adequate [ tile | depth ] key 
	// and corresponding dublicated Gaussian indices to be sorted
	duplicateWithKeys << <(P + 255) / 256, 256 >> > (
		P,
		geomState.means2D,
		geomState.depths,
		geomState.point_offsets,
		binningState.point_list_keys_unsorted,
		binningState.point_list_unsorted,
		radii,
		tile_grid)
	CHECK_CUDA(, debug)

	int bit = getHigherMsb(tile_grid.x * tile_grid.y);//返回最高位的位置，这样就确定了排序瓦片所需要的bit数

	// Sort complete list of (duplicated) Gaussian indices by keys
	CHECK_CUDA(hipcub::DeviceRadixSort::SortPairs(
		binningState.list_sorting_space,
		binningState.sorting_size,
		binningState.point_list_keys_unsorted, binningState.point_list_keys,
		binningState.point_list_unsorted, binningState.point_list,
		num_rendered, 0, 32 + bit), debug)
		//按照后32位排序，也就是按照深度排序，排序号存在point_list_keys和point_list中
		//通过位操作，实现先由瓦片排序，再由深度排序

	CHECK_CUDA(hipMemset(imgState.ranges, 0, tile_grid.x * tile_grid.y * sizeof(uint2)), debug);//将空间先都置0

	// Identify start and end of per-tile workloads in sorted list
	if (num_rendered > 0)
		identifyTileRanges << <(num_rendered + 255) / 256, 256 >> > (
			num_rendered,
			binningState.point_list_keys,
			imgState.ranges);
	CHECK_CUDA(, debug)

	// Let each tile blend its range of Gaussians independently in parallel
	const float* feature_ptr = colors_precomp != nullptr ? colors_precomp : geomState.rgb;
	CHECK_CUDA(FORWARD::render(
		tile_grid, block,
		imgState.ranges,
		binningState.point_list,
		width, height,
		geomState.means2D,
		feature_ptr,
		geomState.conic_opacity,
		imgState.accum_alpha,
		imgState.n_contrib,
		background,
		out_color), debug)

	return num_rendered;
}

// Produce necessary gradients for optimization, corresponding
// to forward render pass
void CudaRasterizer::Rasterizer::backward(
	const int P, int D, int M, int R,
	const float* background,
	const int width, int height,
	const float* means3D,
	const float* shs,
	const float* colors_precomp,
	const float* scales,
	const float scale_modifier,
	const float* rotations,
	const float* cov3D_precomp,
	const float* viewmatrix,
	const float* projmatrix,
	const float* campos,
	const float tan_fovx, float tan_fovy,
	const int* radii,
	char* geom_buffer,
	char* binning_buffer,
	char* img_buffer,
	const float* dL_dpix,
	float* dL_dmean2D,
	float* dL_dconic,
	float* dL_dopacity,
	float* dL_dcolor,
	float* dL_dmean3D,
	float* dL_dcov3D,
	float* dL_dsh,
	float* dL_dscale,
	float* dL_drot,
	bool debug)
{
	GeometryState geomState = GeometryState::fromChunk(geom_buffer, P);
	BinningState binningState = BinningState::fromChunk(binning_buffer, R);
	ImageState imgState = ImageState::fromChunk(img_buffer, width * height);

	if (radii == nullptr)
	{
		radii = geomState.internal_radii;
	}

	const float focal_y = height / (2.0f * tan_fovy);
	const float focal_x = width / (2.0f * tan_fovx);

	const dim3 tile_grid((width + BLOCK_X - 1) / BLOCK_X, (height + BLOCK_Y - 1) / BLOCK_Y, 1);
	const dim3 block(BLOCK_X, BLOCK_Y, 1);

	// Compute loss gradients w.r.t. 2D mean position, conic matrix,
	// opacity and RGB of Gaussians from per-pixel loss gradients.
	// If we were given precomputed colors and not SHs, use them.
	const float* color_ptr = (colors_precomp != nullptr) ? colors_precomp : geomState.rgb;
	CHECK_CUDA(BACKWARD::render(
		tile_grid,
		block,
		imgState.ranges,
		binningState.point_list,
		width, height,
		background,
		geomState.means2D,
		geomState.conic_opacity,
		color_ptr,
		imgState.accum_alpha,
		imgState.n_contrib,
		dL_dpix,
		(float3*)dL_dmean2D,
		(float4*)dL_dconic,
		dL_dopacity,
		dL_dcolor), debug)

	// Take care of the rest of preprocessing. Was the precomputed covariance
	// given to us or a scales/rot pair? If precomputed, pass that. If not,
	// use the one we computed ourselves.
	const float* cov3D_ptr = (cov3D_precomp != nullptr) ? cov3D_precomp : geomState.cov3D;
	CHECK_CUDA(BACKWARD::preprocess(P, D, M,
		(float3*)means3D,
		radii,
		shs,
		geomState.clamped,
		(glm::vec3*)scales,
		(glm::vec4*)rotations,
		scale_modifier,
		cov3D_ptr,
		viewmatrix,
		projmatrix,
		focal_x, focal_y,
		tan_fovx, tan_fovy,
		(glm::vec3*)campos,
		(float3*)dL_dmean2D,
		dL_dconic,
		(glm::vec3*)dL_dmean3D,
		dL_dcolor,
		dL_dcov3D,
		dL_dsh,
		(glm::vec3*)dL_dscale,
		(glm::vec4*)dL_drot), debug)
}